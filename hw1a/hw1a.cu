#include "hip/hip_runtime.h"


#include <stdio.h>
#include <hip/hip_runtime.h>

// STUDENTS: be sure to set the single define at the top of this file, 
// depending on which machines you are running on.
#include "im1.h"



// handy error macro:
#define GPU_CHECKERROR( err ) (gpuCheckError( err, __FILE__, __LINE__ ))
static void gpuCheckError( hipError_t err,
                          const char *file,
                          int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
               file, line );
        exit( EXIT_FAILURE );
    }
}




//
// your __global__ kernel can go here, if you want:
//


int main (int argc, char *argv[])
{
 

    printf("reading openEXR file %s\n", argv[1]);
        
    int w, h;   // the width & height of the image, used frequently!


    // First, convert the openEXR file into a form we can use on the CPU
    // and the GPU: a flat array of floats:
    // This makes an array h*w*sizeof(float)*3, with sequential r/g/b indices
    // don't forget to free it at the end


    float *h_imageArray;
    readOpenEXRFile (argv[1], &h_imageArray, w, h);

    // 
    // serial code: saves the image in "hw1_serial.exr"
    //

    // for every pixel in p, get it's Rgba structure, and convert the
    // red/green/blue values there to luminance L, effectively converting
    // it to greyscale:

    for (int y = 0; y < h; ++y) {
        for (int x = 0; x < w; ++x) {
            
            unsigned int idx = ((y * w) + x) * 3;
            
            float L = 0.2126f*h_imageArray[idx] + 
                      0.7152f*h_imageArray[idx+1] + 
                      0.0722f*h_imageArray[idx+2];

            h_imageArray[idx] = L;
            h_imageArray[idx+1] = L;
            h_imageArray[idx+2] = L;

       }
    }
    
    printf("writing output image hw1_serial.exr\n");
    writeOpenEXRFile ("hw1_serial.exr", h_imageArray, w, h);
    free(h_imageArray); // make sure you free it: if you use this variable
                        // again, readOpenEXRFile will allocate more memory


    //
    // Now the GPU version: it will save whatever is in h_imageArray
    // to the file "hw1_gpu.exr"
    //
    
    // read the file again - the file read allocates memory for h_imageArray:
    readOpenEXRFile (argv[1], &h_imageArray, w, h);



    // at this point, h_imageArray has sequenial floats for red, green , and
    // blue for each pixel: r,g,b,r,g,b,r,g,b,r,g,b. You need to copy
    // this array to GPU global memory, and have one thread per pixel compute
    // the luminance value, with which you will overwrite each r,g,b, triple.

    //
    // process it on the GPU: 1) copy it to device memory, 2) process
    // it with a 2d grid of 2d blocks, with each thread assigned to a 
    // pixel. then 3) copy it back.
    //





    //
    // Your memory copy, & kernel launch code goes here:
    //




    // All your work is done. Here we assume that you have copied the 
    // processed image data back, frmm the device to the host, into the
    // original host array h_imageArray. You can do it some other way,
    // this is just a suggestion
    
    printf("writing output image hw1_gpu.exr\n");
    writeOpenEXRFile ("hw1_gpu.exr", h_imageArray, w, h);
    free (h_imageArray);

    printf("done.\n");

    return 0;
}


